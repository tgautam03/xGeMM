#include "hip/hip_runtime.h"
#include "../include/MatrixFP32.cuh"
#include <assert.h>

#include <iostream>

// Coalescing Factor
#define COARSE_FACTOR 8

// Tiles of A
#define tiles_A_rows 64
#define tiles_A_cols 8

// Tiles of B
#define tiles_B_cols 64

__global__ void coarse_1d_mat_mul_kernel(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr, int C_n_rows, int C_n_cols, int A_n_cols)
{
  // Details regarding this thread
  const int by = blockIdx.y;
  const int bx = blockIdx.x; 

  const int tx = threadIdx.x; 

  // 1D -> 2D while loading A
  const int A_view_ty = tx / tiles_A_cols;
  const int A_view_tx = tx % tiles_A_cols;

  // 1D -> 2D while loading B
  const int B_view_ty = tx / tiles_B_cols;
  const int B_view_tx = tx % tiles_B_cols;

  // Working on C[row,col]
  const int row = tiles_A_rows*by + COARSE_FACTOR * (tx/tiles_B_cols);
  const int col = tiles_B_cols*bx + (tx % tiles_B_cols);

  // Allocating shared memory
  __shared__ float sh_A[tiles_A_rows][tiles_A_cols];
  __shared__ float sh_B[tiles_A_cols][tiles_B_cols];

  // Phases
  const int phases = ceil((float)A_n_cols/tiles_A_cols);

  // Parallel mat mul
  float value[COARSE_FACTOR] = {0.0f};
  for (int phase = 0; phase < phases; phase++)
  {
    // Load Tiles into shared memory
    if ((by*tiles_A_rows + A_view_ty < C_n_rows) && ((phase*tiles_A_cols+A_view_tx) < A_n_cols))
      sh_A[A_view_ty][A_view_tx] = d_A_ptr[(by*tiles_A_rows + A_view_ty)*A_n_cols + (phase*tiles_A_cols+A_view_tx)];
    else
      sh_A[A_view_ty][A_view_tx] = 0.0f;
    
    if (((phase*tiles_A_cols + B_view_ty) < A_n_cols) && (bx*tiles_B_cols + B_view_tx < C_n_cols))
      sh_B[B_view_ty][B_view_tx] = d_B_ptr[(phase*tiles_A_cols + B_view_ty)*C_n_cols + (bx*tiles_B_cols + B_view_tx)];
    else
      sh_B[B_view_ty][B_view_tx] = 0.0f;
    __syncthreads();

    for (int k = 0; k < tiles_A_cols; k++)
    {
      float B_val_register = sh_B[k][B_view_tx];
      // Dot product
      for (int c = 0; c < COARSE_FACTOR; c++)
        value[c] += sh_A[B_view_ty*COARSE_FACTOR+c][k] * B_val_register;  
    }
    __syncthreads();
  }

  // Assigning calculated value
  for (int c = 0; c < COARSE_FACTOR; ++c)
  {
    if ((row+c < C_n_rows) && (col < C_n_cols))
      d_C_ptr[(row+c)*C_n_cols + (col)] = 1*value[c] + 0*d_C_ptr[(row+c)*C_n_cols + (col)];
  } 
}

void coarse_1d_xgemm(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr, int C_n_rows, int C_n_cols, int A_n_cols)
{
  // Kernel execution
  dim3 dim_grid(ceil(C_n_cols/(float)(tiles_B_cols)), ceil(C_n_rows/(float)(tiles_A_rows)));
  dim3 dim_block(tiles_A_rows*tiles_B_cols/COARSE_FACTOR);
  coarse_1d_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A_ptr, d_B_ptr, d_C_ptr, C_n_rows, C_n_cols, A_n_cols);
}