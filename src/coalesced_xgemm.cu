#include "hip/hip_runtime.h"
#include "../include/MatrixFP32.cuh"
#include <assert.h>

__global__ void coalesced_mat_mul_kernel(MatrixFP32 d_A, MatrixFP32 d_B, MatrixFP32 d_C)
{
    // Working on C[row,col]
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    int row = blockDim.y*blockIdx.y + threadIdx.y;

    // Parallel mat mul
    if (row < d_C.rows() && col < d_C.cols())
    {
        // Value at C[row,col]
        float value = 0;
        for (int k = 0; k < d_B.rows(); k++)
        {
            value += d_A.get_val(row, k) * d_B.get_val(k, col);
        }

        // Assigning calculated value (SGEMM is C = α*(A @ B)+β*C and in this repo α=1, β=0)
        d_C.set_val(row, col, 1*value + 0*d_C.get_val(row, col));
    }
}

void coalesced_xgemm(MatrixFP32 d_A, MatrixFP32 d_B, MatrixFP32 d_C, const int dim_block_x, const int dim_block_y)
{
    // Make sure that matirces are on Device
    assert(d_A._on_device == true && "Matrix must be on device");
    assert(d_B._on_device == true && "Matrix must be on device");
    assert(d_C._on_device == true && "Matrix must be on device");

    // Kernel execution
    dim3 dim_block(dim_block_x, dim_block_y, 1);
    dim3 dim_grid(ceil(d_C.cols()/(float)(dim_block_x)), ceil(d_C.rows()/(float)(dim_block_y)), 1);
    coalesced_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C);
}