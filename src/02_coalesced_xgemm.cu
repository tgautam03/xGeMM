#include "hip/hip_runtime.h"
#include "../include/MatrixFP32.cuh"
#include <assert.h>

__global__ void coalesced_mat_mul_kernel(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr, int C_n_rows, int C_n_cols, int A_n_cols)
{
    // Working on C[row,col]
    const int col = blockDim.x*blockIdx.x + threadIdx.x;
    const int row = blockDim.y*blockIdx.y + threadIdx.y;

    // Parallel mat mul
    if (row < C_n_rows && col < C_n_cols)
    {
        // Value at C[row,col]
        float value = 0;
        for (int k = 0; k < A_n_cols; k++)
        {
            value += d_A_ptr[row*A_n_cols + k] * d_B_ptr[k*C_n_cols + col];
        }

        // Assigning calculated value (SGEMM is C = α*(A @ B)+β*C and in this repo α=1, β=0)
        d_C_ptr[row*C_n_cols + col] = 1*value + 0*d_C_ptr[row*C_n_cols + col];
    }
}

void coalesced_xgemm(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr, int C_n_rows, int C_n_cols, int A_n_cols)
{
    // Kernel execution
    dim3 dim_block(32, 32, 1);
    dim3 dim_grid(ceil(C_n_cols/(float)(32)), ceil(C_n_rows/(float)(32)), 1);
    coalesced_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A_ptr, d_B_ptr, d_C_ptr, C_n_rows, C_n_cols, A_n_cols);
}