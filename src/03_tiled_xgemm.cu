#include "hip/hip_runtime.h"
#include "../include/MatrixFP32.cuh"
#include <assert.h>

#define TILE_WIDTH 32

__global__ void tiled_mat_mul_kernel(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr, int C_n_rows, int C_n_cols, int A_n_cols)
{
    // Ensure that TILE_WIDTH = BLOCK_SIZE
    assert(TILE_WIDTH == blockDim.x);
    assert(TILE_WIDTH == blockDim.y);
    
    // Details regarding this thread
    const int by = blockIdx.y;
    const int bx = blockIdx.x; 

    const int ty = threadIdx.y;
    const int tx = threadIdx.x; 

    // Working on C[row,col]
    const int row = TILE_WIDTH*by + ty;
    const int col = TILE_WIDTH*bx + tx;

    // Allocating shared memory
    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    // Phases
    const int phases = ceil((float)A_n_cols/TILE_WIDTH);

    // Parallel mat mul
    float value = 0;
    for (int phase = 0; phase < phases; phase++)
    {
        // Load Tiles into shared memory
        if ((row < C_n_rows) && ((phase*TILE_WIDTH+tx) < A_n_cols))
          sh_A[ty][tx] = d_A_ptr[(row)*A_n_cols + (phase*TILE_WIDTH+tx)];
        else
          sh_A[ty][tx] = 0.0f;

        if (((phase*TILE_WIDTH + ty) < A_n_cols) && (col < C_n_cols))
          sh_B[ty][tx] = d_B_ptr[(phase*TILE_WIDTH + ty)*C_n_cols + (col)];
        else
          sh_B[ty][tx] = 0.0f;
        __syncthreads();

        // Dot product
        for (int k_phase = 0; k_phase < TILE_WIDTH; k_phase++)
            value += sh_A[ty][k_phase] * sh_B[k_phase][tx];
        __syncthreads();
    }
    // Assigning calculated value
    if ((row < C_n_rows) && (col < C_n_cols))
        d_C_ptr[(row)*C_n_cols + (col)] =  1*value + 0*d_C_ptr[(row)*C_n_cols + (col)];
}

void tiled_xgemm(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr, int C_n_rows, int C_n_cols, int A_n_cols)
{
    // Kernel execution
    dim3 dim_block(32, 32, 1);
    dim3 dim_grid(ceil(C_n_cols/(float)(32)), ceil(C_n_rows/(float)(32)), 1);
    tiled_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A_ptr, d_B_ptr, d_C_ptr, C_n_rows, C_n_cols, A_n_cols);
}