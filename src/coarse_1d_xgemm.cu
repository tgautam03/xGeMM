#include "hip/hip_runtime.h"
#include "../include/MatrixFP32.cuh"
#include <assert.h>

#include <stdio.h>

// Coalescing Factor
#define COARSE_FACTOR 2

// Tiles of A
#define tiles_A_rows 4
#define tiles_A_cols 2

// Tiles of B
#define tiles_B_cols 4

__global__ void coarse_1d_mat_mul_kernel(MatrixFP32 d_A, MatrixFP32 d_B, MatrixFP32 d_C)
{
  // Details regarding this thread
  int by = blockIdx.y;
  int bx = blockIdx.x; 

  int tx = threadIdx.x; 

  // 1D -> 2D while loading A
  int A_view_ty = tx / tiles_A_cols;
  int A_view_tx = tx % tiles_A_cols;

  // 1D -> 2D while loading B or solving for C
  int view_ty = tx / tiles_B_cols;
  int view_tx = tx % tiles_B_cols;

  // Working on C[row,col]
  int row = COARSE_FACTOR*tiles_A_rows*by + view_ty;
  int col = tiles_B_cols*bx + view_tx;

  // Allocating shared memory
  __shared__ float sh_A[tiles_A_rows][tiles_A_cols];
  __shared__ float sh_B[tiles_A_cols][tiles_B_cols];

  // Parallel mat mul
  float value[COARSE_FACTOR] = {0.0f};
  for (int phase = 0; phase < ceil((float)d_A.cols()/tiles_A_cols); phase++)
  {
    // Load Tiles into shared memory
    if ((row < d_A.rows()) && ((phase*tiles_A_cols+A_view_tx) < d_A.cols()))
      sh_A[A_view_ty][A_view_tx] = d_A.get_val(row, phase*tiles_A_cols+A_view_tx);
    else
      sh_A[A_view_ty][A_view_tx] = 0.0f;
    
    if (((phase*tiles_A_cols + view_ty) < d_B.rows()) && (col < d_B.cols()))
      sh_B[view_ty][view_tx] = d_B.get_val(phase*tiles_A_cols + view_ty, col);
    else
      sh_B[view_ty][view_tx] = 0.0f;
    __syncthreads();
    
    for (int k = 0; k < tiles_A_cols; k++)
    {
      float B_val_register = sh_B[k][view_tx];
      // Dot product
      for (int c = 0; c < COARSE_FACTOR; c++)
        value[c] += sh_A[view_ty*COARSE_FACTOR+c][k] * B_val_register;  
    }
    __syncthreads();
  }

  // Assigning calculated value
  for (int c = 0; c < COARSE_FACTOR; ++c)
  {
    if ((row + c < d_C.rows()) && (col < d_C.cols()))
      d_C.set_val(row + c, col, 1*value[c] + 0*d_C.get_val(row + c, col));
  } 
}

void coarse_1d_xgemm(MatrixFP32 d_A, MatrixFP32 d_B, MatrixFP32 d_C)
{
  // Kernel execution
  dim3 dim_grid(ceil(d_C.cols()/(float)(tiles_B_cols)), ceil(d_C.rows()/(float)(tiles_A_rows)), 1);
  dim3 dim_block(tiles_A_rows*tiles_B_cols/COARSE_FACTOR, 1);
  coarse_1d_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C);
}