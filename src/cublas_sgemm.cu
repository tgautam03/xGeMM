#include <iostream>

#include <hipblas.h>

#include "../include/cublas_sgemm.cuh"

// CUDA Error Checking
#define cuda_check(err) { \
    if (err != hipSuccess) { \
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << "\n"; \
        exit(EXIT_FAILURE); \
    } \
}

// CUBLAS Error Checking
#define cublas_check(status) { \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error" << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

void cublas_sgemm(float* A, float* B, float* C, int Nrows_A, int Nrows_B, int Ncols_B)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, Nrows_A*Nrows_B*sizeof(float));
    cuda_check(err_A);

    hipError_t err_B = hipMalloc((void**) &d_B, Nrows_B*Ncols_B*sizeof(float));
    cuda_check(err_B);

    hipError_t err_C = hipMalloc((void**) &d_C, Nrows_A*Ncols_B*sizeof(float));
    cuda_check(err_C);

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, Nrows_A*Nrows_B*sizeof(float), hipMemcpyHostToDevice);
    cuda_check(err_A_);

    hipError_t err_B_ = hipMemcpy(d_B, B, Nrows_B*Ncols_B*sizeof(float), hipMemcpyHostToDevice);
    cuda_check(err_B_);

    // Create and initialize cuBLAS handle
    hipblasHandle_t handle;
    cublas_check(hipblasCreate(&handle));

    // Perform matrix multiplication: C = alpha * A * B + beta * C
    float alpha = 1;
    float beta = 0;
    cublas_check(hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             Ncols_B, Nrows_A, Nrows_B, // Num Cols of C, Num rows of C, Shared dim of A and B
                             &alpha,
                             d_B, Ncols_B, // Num cols of B
                             d_A, Nrows_B, // Num cols of A
                             &beta,
                             d_C, Ncols_B) // Num cols of C
                );

    // Copy back results
    hipError_t err_C_ = hipMemcpy(C, d_C, Nrows_A*Ncols_B*sizeof(float), hipMemcpyDeviceToHost);
    cuda_check(err_C_);

    // Free memory
    cublas_check(hipblasDestroy(handle));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
