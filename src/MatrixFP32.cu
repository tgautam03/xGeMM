#include <assert.h>
#include <iostream>

#include "../include/MatrixFP32.cuh"

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

MatrixFP32::MatrixFP32(int n_rows, int n_cols, bool on_device)
{
    // Assigning Number of rows and cols to provate variables
    _n_rows = n_rows;
    _n_cols = n_cols;

    if (on_device == false)
    {
        // Initialize dynamic array
        _mat = new float[_n_rows*_n_cols];
        // Matrix is in host memory (RAM)
        _on_device = on_device;
    }
    else
    {
        // Allocate device memory
        hipError_t err = hipMalloc((void**) &_mat, n_rows*n_cols*sizeof(float));
        CUDA_CHECK(err);
        // Matrix is in device memory (VRAM)
        _on_device = on_device;
    }
}

void MatrixFP32::free_mat()
{
    if (_on_device == false)
        delete[] _mat;
    else
        hipFree(_mat);
}

__host__ __device__ int MatrixFP32::rows() const
{
    return _n_rows;
}

__host__ __device__ int MatrixFP32::cols() const
{
    return _n_cols;
}

__host__ __device__ float MatrixFP32::get_val(int row, int col) const
{
    return _mat[row*_n_cols + col];
}


__host__ __device__ void MatrixFP32::set_val(int row, int col, float val)
{
    _mat[row*_n_cols + col] = val;
}

MatrixFP32 MatrixFP32::copy_to_device()
{
    // Make sure that _mat is on host 
    assert(_on_device == false && "Matrix must be in host memory");

    // Initialize Device Matrix
    MatrixFP32 d_mat(_n_rows, _n_cols, true);

    // Copying from host to device memory
    hipError_t err = hipMemcpy(d_mat._mat, _mat, _n_rows*_n_cols*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err);

    return d_mat;
}

void MatrixFP32::copy_to_host(MatrixFP32 h_mat)
{
    // Make sure that _mat is on device
    assert(_on_device == true && "Matrix must be in device memory");
    assert(h_mat._on_device == false && "Input Matrix to this function must be in host memory");

    // Copying from host to device memory
    hipError_t err = hipMemcpy(h_mat._mat, _mat, _n_rows*_n_cols*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err);
}