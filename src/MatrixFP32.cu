#include <assert.h>
#include <iostream>

#include "../include/MatrixFP32.cuh"

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

MatrixFP32::MatrixFP32(int n_rows, int n_cols, bool on_device)
{
    // Assigning Number of rows and cols to provate variables
    _n_rows = n_rows;
    _n_cols = n_cols;

    if (on_device == false)
    {
        // Initialize dynamic array
        _mat = new float[_n_rows*_n_cols];
        // Matrix is in host memory (RAM)
        _on_device = on_device;
    }
    else
    {
        // Allocate device memory
        hipError_t err = hipMalloc((void**) &_mat, n_rows*n_cols*sizeof(float));
        CUDA_CHECK(err);
        // Matrix is in device memory (VRAM)
        _on_device = on_device;
    }
}

void MatrixFP32::free_mat()
{
    if (_on_device == false)
        delete[] _mat;
    else
        hipFree(_mat);
}

int MatrixFP32::rows()
{
    return _n_rows;
}

int MatrixFP32::cols()
{
    return _n_cols;
}

float MatrixFP32::get_val(int row, int col)
{
    return _mat[row*_n_cols + col];
}

void MatrixFP32::set_val(int row, int col, float val)
{
    _mat[row*_n_cols + col] = val;
}

MatrixFP32 MatrixFP32::copy_to_device()
{
    // Make sure that _mat is on host 
    assert(_on_device == false && "Matrix must be in host memory");

    // Initialize Device Matrix
    MatrixFP32 d_mat(_n_rows, _n_cols, true);

    // Copying from host to device memory
    hipError_t err = hipMemcpy(d_mat._mat, _mat, _n_rows*_n_cols*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err);

    return d_mat;
}

MatrixFP32 MatrixFP32::copy_to_host()
{
    // Make sure that _mat is on device
    assert(_on_device == true && "Matrix must be in host memory");

    // Initialize Device Matrix
    MatrixFP32 h_mat(_n_rows, _n_cols, false);

    // Copying from host to device memory
    hipError_t err = hipMemcpy(h_mat._mat, _mat, _n_rows*_n_cols*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err);

    return h_mat;
}

