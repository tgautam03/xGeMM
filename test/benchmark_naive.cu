#include <iostream>
#include <iomanip>
#include <hipblas.h>

#include "../include/MatrixFP32.cuh"
#include "../include/utils.hpp"

#include "../include/naive_xgemm.cuh"

// CUDA Error Checking
#define cuda_check(err) { \
    if (err != hipSuccess) { \
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << "\n"; \
        exit(EXIT_FAILURE); \
    } \
}

// CUBLAS Error Checking
#define cublas_check(status) { \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error" << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

int main(int argc, char const *argv[])
{
    // Options: 8, 16, 32, 64, 128, 256, 512, 1028, 2048, 4096, 8192
    int mat_sizes[] = {128, 256, 512, 1028, 2048, 4096};
    int n_sizes = sizeof(mat_sizes) / sizeof(mat_sizes[0]);

    // For recording time
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // Store time and GFLOPS
    double cublas_time[n_sizes];
    double cublas_gflops[n_sizes];
    double xgemm_time[n_sizes];
    double xgemm_gflops[n_sizes];

    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
    {
        // Matrix Size
        int n = mat_sizes[mat_size];

        // Define MatrixFP32
        MatrixFP32 A_FP32 = MatrixFP32(n, n, false);
        MatrixFP32 B_FP32 = MatrixFP32(n, n, false);
        MatrixFP32 C_FP32_cublas = MatrixFP32(n, n, false);
        MatrixFP32 C_FP32_xgemm = MatrixFP32(n, n, false);

        // Initialize Matrices
        random_init_mat(A_FP32, -10, 10);          // Random Initialization between -10 and 10
        random_init_mat(B_FP32, -10, 10);          // Random Initialization between -10 and 10
        init_mat(C_FP32_cublas, 1.0f);     // Initialize to 1
        init_mat(C_FP32_xgemm, -1.0f);     // Initialize to -1

        // Move matrices to device
        MatrixFP32 d_A_FP32 = MatrixFP32(n, n, true); 
        A_FP32.copy_to_device(d_A_FP32);
        MatrixFP32 d_B_FP32 = MatrixFP32(n, n, true); 
        B_FP32.copy_to_device(d_B_FP32);
        MatrixFP32 d_C_FP32_cublas = MatrixFP32(n, n, true); 
        C_FP32_cublas.copy_to_device(d_C_FP32_cublas);
        MatrixFP32 d_C_FP32_xgemm = MatrixFP32(n, n, true); 
        C_FP32_xgemm.copy_to_device(d_C_FP32_xgemm);
        hipDeviceSynchronize();

        //----------------------------------------------------//
        //-------------------- Warmup Run --------------------//
        //----------------------------------------------------//
        // Create and initialize cuBLAS handle
        hipblasHandle_t handle;
        cublas_check(hipblasCreate(&handle));
        
        // Perform matrix multiplication: C = A * B 
        float alpha = 1;
        float beta = 0;
        cublas_check(hipblasSgemm(handle,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                d_C_FP32_cublas.n_cols, d_C_FP32_cublas.n_rows, d_A_FP32.n_cols, // Num Cols of C, Num rows of C, Shared dim of A and B
                                &alpha,
                                d_B_FP32.ptr, d_B_FP32.n_cols, // Num cols of B
                                d_A_FP32.ptr, d_A_FP32.n_cols, // Num cols of A
                                &beta,
                                d_C_FP32_cublas.ptr, d_C_FP32_cublas.n_cols) // Num cols of C
                    );
        hipDeviceSynchronize();

        // Naive Kernel execution
        naive_xgemm(d_A_FP32, d_B_FP32, d_C_FP32_xgemm);
        hipDeviceSynchronize();

        // Assert that naive implementation is correct
        d_C_FP32_cublas.copy_to_host(C_FP32_cublas);
        d_C_FP32_xgemm.copy_to_host(C_FP32_xgemm);
        std::cout << "Asserting Results for N: " << n << "\n";
        assert_mat(C_FP32_xgemm, C_FP32_cublas, 1e-8);
        std::cout << "Assertion Passed! \n \n";

        // Printing the smallest matrix result
        if (n == 8)
        {
            std::cout << "Matrix C (cuBLAS): \n";
            print_mat(C_FP32_cublas, true);
            std::cout << "\n";

            std::cout << "Matrix C (xGeMM): \n";
            print_mat(C_FP32_xgemm, true);
            std::cout << "\n";
        }

        //----------------------------------------------------//
        //--------------------- cuBLAS -----------------------//
        //----------------------------------------------------//
        hipEventRecord(beg);
        for (int n_runs = 0; n_runs < 10; n_runs++)
        {
            float alpha = 1;
            float beta = 0;
            cublas_check(hipblasSgemm(handle,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                d_C_FP32_cublas.n_cols, d_C_FP32_cublas.n_rows, d_A_FP32.n_cols, // Num Cols of C, Num rows of C, Shared dim of A and B
                                &alpha,
                                d_B_FP32.ptr, d_B_FP32.n_cols, // Num cols of B
                                d_A_FP32.ptr, d_A_FP32.n_cols, // Num cols of A
                                &beta,
                                d_C_FP32_cublas.ptr, d_C_FP32_cublas.n_cols) // Num cols of C
                    );
            hipDeviceSynchronize();
        }
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.;

        cublas_time[mat_size] = (elapsed_time) / 10;
        cublas_gflops[mat_size] = 2. * 1e-9 * 10 * n * n * n / (elapsed_time);

        //----------------------------------------------------//
        //---------------------- xGeMM -----------------------//
        //----------------------------------------------------//
        hipEventRecord(beg);
        for (int n_runs = 0; n_runs < 10; n_runs++)
        {
            naive_xgemm(d_A_FP32, d_B_FP32, d_C_FP32_xgemm);
            hipDeviceSynchronize();
        }
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.;

        xgemm_time[mat_size] = (elapsed_time) / 10;
        xgemm_gflops[mat_size] = 2. * 1e-9 * 10 * n * n * n / (elapsed_time);

        // Free Memory
        A_FP32.free_mat();
        B_FP32.free_mat();
        C_FP32_cublas.free_mat();
        C_FP32_xgemm.free_mat();

        d_A_FP32.free_mat();
        d_B_FP32.free_mat();
        d_C_FP32_cublas.free_mat();
        d_C_FP32_xgemm.free_mat();
    }

    std::cout << "Matrix Size: ";
    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
        std::cout << mat_sizes[mat_size] << " ";
    std::cout << "\n \n";

    std::cout << "cuBLAS Time (seconds): ";
    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
        std::cout << cublas_time[mat_size] << " ";
    std::cout << "\n";
    std::cout << "xGeMM Time (seconds): ";
    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
        std::cout << xgemm_time[mat_size] << " ";
    std::cout << "\n \n";

    std::cout << "cuBLAS GFLOPS: ";
    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
        std::cout << cublas_gflops[mat_size] << " ";
    std::cout << "\n";
    std::cout << "xGeMM GFLOPS: ";
    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
        std::cout << xgemm_gflops[mat_size] << " ";
    std::cout << "\n \n";

    std::cout << "cuBLAS vs Naive xGeMM (CuBLAS/xGeMM): ";
    for (int mat_size = 0; mat_size < n_sizes; mat_size++)
        std::cout << std::fixed << std::setprecision(2) << cublas_time[mat_size]/xgemm_time[mat_size]*100 << "% ";
    std::cout << "\n";

    // Saving to benchmark file
    update_benckmark_txt("txt_benchmarks/cublas.txt", cublas_time, cublas_gflops, mat_sizes, n_sizes);
    update_benckmark_txt("txt_benchmarks/naive_xgemm.txt", xgemm_time, xgemm_gflops, mat_sizes, n_sizes);

    return 0;
}
